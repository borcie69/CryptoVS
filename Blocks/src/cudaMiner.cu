#include "hip/hip_runtime.h"
#include "cudaMiner.h"

#ifdef CUDA_MINING

#include <hip/hip_runtime.h>

__global__ void kernel()
{
	printf("Hello, World!\n");
}


void cudaMiner::doCuda()
{
	kernel<<<1,1>>>();
	hipDeviceSynchronize();
}

#endif //CUDA_MINING