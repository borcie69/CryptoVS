#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include <iostream>
#include "cudaMiner.h"

__global__ void mykernel()
{
	printf("Hello, World!\n");
}

void cudaMiner::doCuda()
{
	mykernel<<<1,1>>>();
	hipDeviceSynchronize();
}